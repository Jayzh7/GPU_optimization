#include "hip/hip_runtime.h"
#include "preprocessing.h"
#include "logging.h"

//CPU version
void cpu_preprocess(BLOB* img){
    //Subtract mean RGB values, scale with 0.017, and swap RGB->BGR
    for(int y=0;y<img->h;y++)
        for(int x=0;x<img->w;x++){
            float R              =  (blob_data(img,0,y,x)-123.680f)*0.017f; //R
            blob_data(img, 1,y,x) = (blob_data(img,1,y,x)-116.779f)*0.017f; //G
            blob_data(img, 2,y,x) = (blob_data(img,2,y,x)-103.939f)*0.017f; //B
            blob_data(img, 0,y,x) = R;
         }
}


//GPU device code (what the threads execute)
__global__ void gpu_device_preprocess(float* data){
    //This code gets executed by each thread in the GPU
    //First step is identifying which thread we are

    // The ids of this thread within our block (commented here since this particular kernel doesn't need them)
    //unsigned int local_x = threadIdx.x; //x coordinate *within* the block this thread maps to
    //unsigned int local_y = threadIdx.y; //y coordinate *within* the block this thread maps to
    //unsigned int local_z = threadIdx.z; //z coordinate *within* the block this thread maps to

    // The global ids (where is this thread in the total grid)
    unsigned int global_x = blockIdx.x*blockDim.x + threadIdx.x;  //blockid*width_of_a_block + local_x
    unsigned int global_y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int global_z = blockIdx.z*blockDim.z + threadIdx.z; //NOTE: gridDim.z==1 and thus blockIdx.z==0 in this example!

    //The image height and width can be passed as an argument to this kernel, but they
    //can also be derived by using the grid and block dimensions in this case
    unsigned int img_width  = gridDim.x*blockDim.x;
    unsigned int img_height = gridDim.y*blockDim.y;

    //load single pixel from global memory into register
    //HINT: the global memory is very slow, so if you have multiple uses of the same pixel, it might be smart to look into the "shared memory".
    //Here however there is only one use of each pixel, so nothing to be gained from using shared memory
    float value = data[ global_z*img_height*img_width + global_y*img_width + global_x];

    //each channel (Z) needs to correct with a different mean value
    float mean[3]={
        123.680f,
        116.779f,
        103.939f
    };

    //correct by subtracting the correct mean for this channel and scaling by a factor 0.017
    value= (value-mean[global_z]) * 0.017f;

    //time to commit the value to the global memory
    //note that we swap RGB to BGR (2-z), as required by the preprocessing
    data[(2-global_z)*img_width*img_height + global_y*img_width + global_x]=value;
}

//GPU host code (called from the CPU, copies data back and forth and launched the GPU thread)
void gpu_preprocess(BLOB* img){

    //GPU code

    /*
     *  Our high level strategy to perform the preprocessing in parallel in this example is to split each of the 3 channels into a number of blocks (numBlocks).
     *  Each block contains thus channel_length/numBlocks elements
     *  These blocks will be mapped to the Streaming Multiprocessors of the GPU.
     *  For each block the preprocessing is performed
    */

    //let's decide on a number of blocks per channel
    int numBlocksX=16;
    int numBlocksY=16;
    int threadsPerBlockX=img->w/numBlocksX;  //NOTE: this should have remainder==0 !!
    int threadsPerBlockY=img->h/numBlocksY; //NOTE: this should have remainder==0 !!


    info("Grid dimensions %d x %d (x 1)\n", numBlocksX,numBlocksY);
    info("Block dimensions %d x %d x 3\n", threadsPerBlockX,threadsPerBlockY);

    dim3 grid( numBlocksX, numBlocksY, 1 ); // numBlocksX x numBlocksY ( x 1)
    dim3 block(threadsPerBlockX, threadsPerBlockY, 3); // threadsPerBlockX x threadsPerBlockY x 3

    //to save on some transfer overhead, the image data is flattened into a 1D array on the GPU
    //first allocate the space of 3 complete channels (Height x Width x Depth)

    //pointer to data on GPU
    float* device_data;

    //variable for holding return values of cuda functions
    hipError_t err;

    //malloc on the GPU
    err=hipMalloc(&device_data, blob_bytes(img));

    //check for errors (NOTE: this is not a standard cuda function. Check logging.h)
    cudaCheckError(err)

    //copy the data over to the GPU
    cudaCheckError(hipMemcpy(device_data, img->data, blob_bytes(img), hipMemcpyHostToDevice));

    //Perform the preprocessing on the GPU
    info("Preprocessing on GPU...\n");
    gpu_device_preprocess<<< grid, block >>>(device_data);

    //We use "peekatlasterror" since a kernel launch does not return a hipError_t to check for errors
    cudaCheckError(hipPeekAtLastError());

    //copy the processed image data back from GPU global memory to CPU memory
    cudaCheckError(hipMemcpy(img->data, device_data, blob_bytes(img), hipMemcpyDeviceToHost));

    //free the allocated GPU memory for this channel
    cudaCheckError(hipFree(device_data));
}
