#include "hip/hip_runtime.h"
#include "logging.h"
#include "image_util.h"

//CPU version
void cpu_preprocess(IMG* img){
    //Subtract mean RGB values, scale with 0.017, and swap RGB->BGR
    for(int y=0;y<img->height;y++)
        for(int x=0;x<img->width;x++){
            float R            = (img->data[0][y][x]-123.680f)*0.017f; //R
            img->data[1][y][x] = (img->data[1][y][x]-116.779f)*0.017f; //G
            img->data[0][y][x] = (img->data[2][y][x]-103.939f)*0.017f; //B
            img->data[2][y][x] = R;
         }
}


//GPU device code (what the threads execute)
__global__ void gpu_device_preprocess(float** channel_data, int img_width){
    //This code gets executed by each thread in the GPU
    //First step is identifying which thread we are

    // The ids of this thread within our block
    //unsigned int local_x = threadIdx.x; //x coordinate *within* the block this thread maps to
    //unsigned int local_y = threadIdx.y; //y coordinate *within* the block this thread maps to
    //unsigned int local_z = threadIdx.z; //z coordinate *within* the block this thread maps to

    // The global ids
    unsigned int global_x = blockIdx.x*blockDim.x + threadIdx.x;  //blockid*width_of_a_block + local_x
    unsigned int global_y = blockIdx.y*blockDim.y + threadIdx.y;
    //note that global_z==local_z! The grid of blocks is only 2D (x and y), so no blocks in the channel dimension!
    unsigned int global_z = threadIdx.z;

    //load single pixel from global memory into register
    //HINT: the global memory is very slow, so if you have multiple uses of the same pixel, it might be smart to look into the "shared memory".
    //Here however there is only one use of each pixel, so nothing to be gained from using shared memory
    //recall that the XxY arrays were flattend to a 1D array, so we have to do our own address calculations to get to the right pixel;
    float value = channel_data[global_z][global_y*img_width+global_x];

    //each channel (Z) needs to correct with a different value
    float mean[3]={
        123.680f,
        116.779f,
        103.939f
    };

    //correct by subtracting the correct mean for this channel and scaling by the factor 0.017
    value= (value-mean[global_z]) * 0.017f;

    //time to commit the value to the global memory
    //note that we swap RGB to BGR, as required by the preprocessing, by inverting bit 1
    unsigned int z = (global_z&(~0x2))|(~(global_z&0x2));
    channel_data[z][global_y*img_width+global_z] = value;
}

//GPU host code (called from the CPU, copies data back and forth and launched the GPU thread)
void gpu_preprocess(IMG* img){

    //GPU code

    /*
     *  Our high level strategy to perform the preprocessing in parallel in this example is to split each of the 3 channels into a number of blocks (numBlocks).
     *  Each block contains thus channel_length/numBlocks elements
     *  These blocks will be mapped to the Streaming Multiprocessors of the GPU.
     *  For each block the preprocessing is performed
    */

    //let's decide on a number of blocks per channel
    int numBlocksX=4;
    int numBlocksY=4;
    int threadsPerBlockX=img->width/numBlocksX; //NOTE: this should have remainder==0 !!
    int threadsPerBlockY=img->width/numBlocksY; //NOTE: this should have remainder==0 !!

    dim3 grid( numBlocksX, numBlocksY ); // numBlocksX x numBlocksY ( x 1)
    dim3 block(threadsPerBlockX, threadsPerBlockY, 3  ); // threadsPerBlockX x threadsPerBlockY x 3

    //Allocate 3 channels on the global memory of the GPU, and transfer data from CPU to GPU
    float* device_channels[3];
    for(int c=0;c<3;c++){

        //variable for holding return values of cuda functions
        hipError_t err;

        //allocate the space of 1 complete channel (Height x Width)
        err=hipMalloc(&device_channels[c], img->width*img->height*sizeof(float));

        //Here we check for errors of this cuda call
        //See logging.h for the implementation of this error check (it's not a default cuda function)
        cudaCheckError(err);

        //Now copy the channel from the CPU to the GPU (note: now directly wrapped in the error checking function)
        //also, the XY grid is flattend into a single 1D grid so only one hipMalloc is needed per channel.
        //channels could of course also be flattened, but for the example we won't do that
        for(int y=0;y<img->height;y++)
            cudaCheckError(hipMemcpy(&(device_channels[c][y*img->width]), img->data[c][y], img->width*sizeof(float), hipMemcpyHostToDevice));
    }

    //Perform the preprocessing on the GPU
    gpu_device_preprocess<<< grid, block >>>(device_channels, img->width);

    //We use "peekatlasterror" since a kernel launch does not return a hipError_t to check for errors
    cudaCheckError(hipPeekAtLastError());

    //copy the processed image data back from GPU global memory to CPU memory
    for(int c=0;c<3;c++){
        for(int y=0;y<img->height;y++)
            cudaCheckError(hipMemcpy(img->data[c][y], &(device_channels[c][y*img->width]), img->width*sizeof(float), hipMemcpyDeviceToHost));

        //free the allocate GPU memory
        cudaCheckError(hipFree(device_channels[c]));
    }
}
